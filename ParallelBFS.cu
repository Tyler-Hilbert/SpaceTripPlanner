#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <time.h>
#define NUM_NODES 34
#define NUM_EDGES 272

#define CUDACHECKERROR( call ) {			\
	hipError_t result = call;              \
	if ( hipSuccess != result )            \
    fprintf(stderr, "CUDA error %i in %s:%s\n%s", result, __FILE__, __LINE__, hipGetErrorString( result ));  \
}

typedef struct
{
	int start;					// Starting index of edges
	int length;					// Number of edges
	char id[2] = {'0', '0'};	// Text ID of node, for display
} Node;

__global__ void Bfs_Kernel(Node* vertexList, int* edgeList, int* frontierList, bool* visitedList, int* distanceList, bool* complete, char* pathList)
{
	// Calculate vertix ID from threads/blocks
	int id = threadIdx.x;

	if (frontierList[id] == 1 && visitedList[id] == false)
	{
		// Check off node in visited array and frontier array
		visitedList[id] = true;
		frontierList[id] = 0;
		__syncthreads();

		// Calculate span of edges to process for the node
		int start = vertexList[id].start;
		int end = start + vertexList[id].length;
		
		// Process Edges
		for (int i = start; i < end; i++)
		{
			// Pull edge vertix id
			int childID = edgeList[i];
			
			// If neighbor vertex has not been visited
			if (visitedList[childID] == false && (atomicCAS(&frontierList[childID], 0, 1) != 1))
			{
				// Copy ID list to neighbor vertex
				for (int j = 0; j < NUM_NODES * 2; ++j) {
					pathList[childID * NUM_NODES * 2 + j] = pathList[id * NUM_NODES * 2 + j];
				}

				// Update vertex distance
				distanceList[childID] = distanceList[id] + 1;

				// Copy newest vertex ID
				pathList[childID * NUM_NODES * 2 + distanceList[childID]*2] = vertexList[id].id[0];
				pathList[childID * NUM_NODES * 2 + distanceList[childID]*2 + 1] = vertexList[id].id[1];

				// Set neighbor to true for frontier array
				*complete = false;
			}

		}

	}

}



int main(){

	// INIT GRAPH DATA
	Node node[NUM_NODES];
	int edges[NUM_EDGES];

	node[00].start = 0;
	node[00].length = 9;
	node[01].start = 9;
	node[01].length = 8;
	node[02].start = 17;
	node[02].length = 8;
	node[03].start = 25;
	node[03].length = 8;
	node[04].start = 33;
	node[04].length = 8;
	node[05].start = 41;
	node[05].length = 11;
	node[06].start = 52;
	node[06].length = 3;
	node[07].start = 55;
	node[07].length = 13;
	node[8].start = 68;
	node[8].length = 1;
	node[9].start = 69;
	node[9].length = 4;
	node[10].start = 73;
	node[10].length = 13;
	node[11].start = 86;
	node[11].length = 12;
	node[12].start = 98;
	node[12].length = 13;
	node[13].start = 111;
	node[13].length = 1;
	node[14].start = 112;
	node[14].length = 12;
	node[15].start = 124;
	node[15].length = 3;
	node[16].start = 127;
	node[16].length = 13;
	node[17].start = 140;
	node[17].length = 13;
	node[18].start = 153;
	node[18].length = 11;
	node[19].start = 164;
	node[19].length = 11;
	node[20].start = 175;
	node[20].length = 4;
	node[21].start = 179;
	node[21].length = 1;
	node[22].start = 180;
	node[22].length = 2;
	node[23].start = 182;
	node[23].length = 8;
	node[24].start = 190;
	node[24].length = 12;
	node[25].start = 202;
	node[25].length = 8;
	node[26].start = 210;
	node[26].length = 13;
	node[27].start = 223;
	node[27].length = 8;
	node[28].start = 231;
	node[28].length = 2;
	node[29].start = 233;
	node[29].length = 2;
	node[30].start = 235;
	node[30].length = 13;
	node[31].start = 248;
	node[31].length = 8;
	node[32].start = 256;
	node[32].length = 13;
	node[33].start = 269;
	node[33].length = 3;
	node[0].id[0] = '0'; node[0].id[1] = '0';
	node[1].id[0] = '0'; node[1].id[1] = '1';

	node[2].id[0] = '0'; node[2].id[1] = '2';

	node[3].id[0] = '0'; node[3].id[1] = '3';

	node[4].id[0] = '0'; node[4].id[1] = '4';

	node[5].id[0] = '0'; node[5].id[1] = '5';

	node[6].id[0] = '0'; node[6].id[1] = '6';

	node[7].id[0] = '0'; node[7].id[1] = '7';

	node[8].id[0] = '0'; node[8].id[1] = '8';

	node[9].id[0] = '0'; node[9].id[1] = '9';

	node[10].id[0] = '1'; node[10].id[1] = '0';

	node[11].id[0] = '1'; node[11].id[1] = '1';

	node[12].id[0] = '1'; node[12].id[1] = '2';

	node[13].id[0] = '1'; node[13].id[1] = '3';

	node[14].id[0] = '1'; node[14].id[1] = '4';

	node[15].id[0] = '1'; node[15].id[1] = '5';

	node[16].id[0] = '1'; node[16].id[1] = '6';

	node[17].id[0] = '1'; node[17].id[1] = '7';

	node[18].id[0] = '1'; node[18].id[1] = '8';

	node[19].id[0] = '1'; node[19].id[1] = '9';

	node[20].id[0] = '2'; node[20].id[1] = '0';

	node[21].id[0] = '2'; node[21].id[1] = '1';

	node[22].id[0] = '2'; node[22].id[1] = '2';

	node[23].id[0] = '2'; node[23].id[1] = '3';

	node[24].id[0] = '2'; node[24].id[1] = '4';

	node[25].id[0] = '2'; node[25].id[1] = '5';

	node[26].id[0] = '2'; node[26].id[1] = '6';

	node[27].id[0] = '2'; node[27].id[1] = '7';

	node[28].id[0] = '2'; node[28].id[1] = '8';

	node[29].id[0] = '2'; node[29].id[1] = '9';

	node[30].id[0] = '3'; node[30].id[1] = '0';

	node[31].id[0] = '3'; node[31].id[1] = '1';

	node[32].id[0] = '3'; node[32].id[1] = '2';

	node[33].id[0] = '3'; node[33].id[1] = '3';
	edges[0] = 01;
	edges[1] = 02;
	edges[2] = 03;
	edges[3] = 04;
	edges[4] = 23;
	edges[5] = 24;
	edges[6] = 25;
	edges[7] = 27;
	edges[8] = 31;
	edges[9] = 00;
	edges[10] = 02;
	edges[11] = 03;
	edges[12] = 04;
	edges[13] = 23;
	edges[14] = 25;
	edges[15] = 27;
	edges[16] = 31;
	edges[17] = 00;
	edges[18] = 01;
	edges[19] = 03;
	edges[20] = 04;
	edges[21] = 23;
	edges[22] = 24;
	edges[23] = 25;
	edges[24] = 31;
	edges[25] = 00;
	edges[26] = 01;
	edges[27] = 02;
	edges[28] = 04;
	edges[29] = 23;
	edges[30] = 25;
	edges[31] = 27;
	edges[32] = 31;
	edges[33] = 00;
	edges[34] = 01;
	edges[35] = 02;
	edges[36] = 03;
	edges[37] = 23;
	edges[38] = 25;
	edges[39] = 27;
	edges[40] = 31;
	edges[41] = 07;
	edges[42] = 10;
	edges[43] = 11;
	edges[44] = 12;
	edges[45] = 14;
	edges[46] = 16;
	edges[47] = 17;
	edges[48] = 24;
	edges[49] = 26;
	edges[50] = 30;
	edges[51] = 32;
	edges[52] = 9;
	edges[53] = 28;
	edges[54] = 29;
	edges[55] = 05;
	edges[56] = 10;
	edges[57] = 11;
	edges[58] = 12;
	edges[59] = 14;
	edges[60] = 16;
	edges[61] = 17;
	edges[62] = 18;
	edges[63] = 19;
	edges[64] = 24;
	edges[65] = 26;
	edges[66] = 30;
	edges[67] = 32;
	edges[68] = 13;
	edges[69] = 06;
	edges[70] = 15;
	edges[71] = 20;
	edges[72] = 33;
	edges[73] = 05;
	edges[74] = 07;
	edges[75] = 11;
	edges[76] = 12;
	edges[77] = 14;
	edges[78] = 16;
	edges[79] = 17;
	edges[80] = 18;
	edges[81] = 19;
	edges[82] = 24;
	edges[83] = 26;
	edges[84] = 30;
	edges[85] = 32;
	edges[86] = 05;
	edges[87] = 07;
	edges[88] = 10;
	edges[89] = 12;
	edges[90] = 14;
	edges[91] = 16;
	edges[92] = 17;
	edges[93] = 19;
	edges[94] = 24;
	edges[95] = 26;
	edges[96] = 30;
	edges[97] = 32;
	edges[98] = 05;
	edges[99] = 07;
	edges[100] = 10;
	edges[101] = 11;
	edges[102] = 14;
	edges[103] = 16;
	edges[104] = 17;
	edges[105] = 18;
	edges[106] = 19;
	edges[107] = 24;
	edges[108] = 26;
	edges[109] = 30;
	edges[110] = 32;
	edges[111] = 8;
	edges[112] = 05;
	edges[113] = 07;
	edges[114] = 10;
	edges[115] = 11;
	edges[116] = 12;
	edges[117] = 16;
	edges[118] = 17;
	edges[119] = 18;
	edges[120] = 19;
	edges[121] = 26;
	edges[122] = 30;
	edges[123] = 32;
	edges[124] = 9;
	edges[125] = 20;
	edges[126] = 33;
	edges[127] = 05;
	edges[128] = 07;
	edges[129] = 10;
	edges[130] = 11;
	edges[131] = 12;
	edges[132] = 14;
	edges[133] = 17;
	edges[134] = 18;
	edges[135] = 19;
	edges[136] = 24;
	edges[137] = 26;
	edges[138] = 30;
	edges[139] = 32;
	edges[140] = 05;
	edges[141] = 07;
	edges[142] = 10;
	edges[143] = 11;
	edges[144] = 12;
	edges[145] = 14;
	edges[146] = 16;
	edges[147] = 18;
	edges[148] = 19;
	edges[149] = 24;
	edges[150] = 26;
	edges[151] = 30;
	edges[152] = 32;
	edges[153] = 07;
	edges[154] = 10;
	edges[155] = 12;
	edges[156] = 14;
	edges[157] = 16;
	edges[158] = 17;
	edges[159] = 19;
	edges[160] = 20;
	edges[161] = 26;
	edges[162] = 30;
	edges[163] = 32;
	edges[164] = 07;
	edges[165] = 10;
	edges[166] = 11;
	edges[167] = 12;
	edges[168] = 14;
	edges[169] = 16;
	edges[170] = 17;
	edges[171] = 18;
	edges[172] = 26;
	edges[173] = 30;
	edges[174] = 32;
	edges[175] = 9;
	edges[176] = 15;
	edges[177] = 18;
	edges[178] = 33;
	edges[179] = 22;
	edges[180] = 21;
	edges[181] = 27;
	edges[182] = 00;
	edges[183] = 01;
	edges[184] = 02;
	edges[185] = 03;
	edges[186] = 04;
	edges[187] = 25;
	edges[188] = 27;
	edges[189] = 31;
	edges[190] = 00;
	edges[191] = 02;
	edges[192] = 05;
	edges[193] = 07;
	edges[194] = 10;
	edges[195] = 11;
	edges[196] = 12;
	edges[197] = 16;
	edges[198] = 17;
	edges[199] = 26;
	edges[200] = 30;
	edges[201] = 32;
	edges[202] = 00;
	edges[203] = 01;
	edges[204] = 02;
	edges[205] = 03;
	edges[206] = 04;
	edges[207] = 23;
	edges[208] = 27;
	edges[209] = 31;
	edges[210] = 05;
	edges[211] = 07;
	edges[212] = 10;
	edges[213] = 11;
	edges[214] = 12;
	edges[215] = 14;
	edges[216] = 16;
	edges[217] = 17;
	edges[218] = 18;
	edges[219] = 19;
	edges[220] = 24;
	edges[221] = 30;
	edges[222] = 32;
	edges[223] = 00;
	edges[224] = 01;
	edges[225] = 03;
	edges[226] = 04;
	edges[227] = 22;
	edges[228] = 23;
	edges[229] = 25;
	edges[230] = 31;
	edges[231] = 06;
	edges[232] = 29;
	edges[233] = 06;
	edges[234] = 28;
	edges[235] = 05;
	edges[236] = 07;
	edges[237] = 10;
	edges[238] = 11;
	edges[239] = 12;
	edges[240] = 14;
	edges[241] = 16;
	edges[242] = 17;
	edges[243] = 18;
	edges[244] = 19;
	edges[245] = 24;
	edges[246] = 26;
	edges[247] = 32;
	edges[248] = 00;
	edges[249] = 01;
	edges[250] = 02;
	edges[251] = 03;
	edges[252] = 04;
	edges[253] = 23;
	edges[254] = 25;
	edges[255] = 27;
	edges[256] = 05;
	edges[257] = 07;
	edges[258] = 10;
	edges[259] = 11;
	edges[260] = 12;
	edges[261] = 14;
	edges[262] = 16;
	edges[263] = 17;
	edges[264] = 18;
	edges[265] = 19;
	edges[266] = 24;
	edges[267] = 26;
	edges[268] = 30;
	edges[269] = 9;
	edges[270] = 15;
	edges[271] = 20;


	int frontier[NUM_NODES] = { 0 };
	int pathLength[NUM_NODES] = { 0 };
	bool visited[NUM_NODES] = { false };
	char path[NUM_NODES * NUM_NODES * 2] = { '0' };

	// Set source node as first to be visited
	int sourceIdx = 0;
	frontier[sourceIdx] = 1;

	// Init data on device
	Node* vertexList;
	int* edgeList;
	int* frontierList;
	bool* visitedList;
	int* distanceList;
	char* pathList;

	CUDACHECKERROR(hipMalloc((void**)&vertexList, sizeof(Node) * NUM_NODES));
	CUDACHECKERROR(hipMalloc((void**)&edgeList, sizeof(int) * NUM_EDGES));
	CUDACHECKERROR(hipMalloc((void**)&frontierList, sizeof(int) * NUM_NODES));
	CUDACHECKERROR(hipMalloc((void**)&visitedList, sizeof(bool) * NUM_NODES));
	CUDACHECKERROR(hipMalloc((void**)&distanceList, sizeof(int) * NUM_NODES));
	CUDACHECKERROR(hipMalloc((void**)&pathList, sizeof(char) * NUM_NODES * NUM_NODES * 2));

	CUDACHECKERROR(hipMemcpy(vertexList, node, sizeof(Node) * NUM_NODES, hipMemcpyHostToDevice));
	CUDACHECKERROR(hipMemcpy(edgeList, edges, sizeof(int) * NUM_EDGES, hipMemcpyHostToDevice));
	CUDACHECKERROR(hipMemcpy(frontierList, frontier, sizeof(int) * NUM_NODES, hipMemcpyHostToDevice));
	CUDACHECKERROR(hipMemcpy(visitedList, visited, sizeof(bool) * NUM_NODES, hipMemcpyHostToDevice));
	CUDACHECKERROR(hipMemcpy(distanceList, pathLength, sizeof(int) * NUM_NODES, hipMemcpyHostToDevice));
	CUDACHECKERROR(hipMemcpy(pathList, path, sizeof(char) * NUM_NODES * NUM_NODES * 2, hipMemcpyHostToDevice));

	// Prepare for loop
	bool complete;
	bool* completeDevice;
	CUDACHECKERROR(hipMalloc((void**)&completeDevice, sizeof(bool)));
	int count = 0;

	// Setup timing
	float time;
	hipEvent_t start, stop;

	// Start timer
	CUDACHECKERROR(hipEventCreate(&start));
	CUDACHECKERROR(hipEventCreate(&stop));
	CUDACHECKERROR(hipEventRecord(start, 0));

	// Start BFS
	do {
		complete = true;
		CUDACHECKERROR(hipMemcpy(completeDevice, &complete, sizeof(bool), hipMemcpyHostToDevice));

		// While kernel not complete, loop until complete
		Bfs_Kernel << <1, NUM_NODES >> > (vertexList, edgeList, frontierList, visitedList, distanceList, completeDevice, pathList);

		// Get current result from device
		CUDACHECKERROR(hipMemcpy(&complete, completeDevice, sizeof(bool), hipMemcpyDeviceToHost));
		count++;

	} while (!complete);

	// End timer
	CUDACHECKERROR(hipEventRecord(stop, 0));
	CUDACHECKERROR(hipEventSynchronize(stop));
	CUDACHECKERROR(hipEventElapsedTime(&time, start, stop));


	// Copy results to host
	CUDACHECKERROR(hipMemcpy(pathLength, distanceList, sizeof(int) * NUM_NODES, hipMemcpyDeviceToHost));
	CUDACHECKERROR(hipMemcpy(path, pathList, sizeof(char) * NUM_NODES * NUM_NODES * 2, hipMemcpyDeviceToHost));

	// print results
	printf("\n------\n");
	char a, b;
	for (int i = 0; i < NUM_NODES; ++i) {
		printf("%d:\t", i);
		for (int j = 0; j < NUM_NODES; ++j){
			a = path[i * NUM_NODES * 2 + j * 2];
			b = path[i * NUM_NODES * 2 + j * 2 + 1];
			if(a >= '0' && a <= '9' && b >= '0' && b <= '9')
				printf("%c%c,  ", a, b);
		}
		printf("\n");
	}
	printf("------\n");


	printf("\nTimes kernel was called: %d\n", count);
	printf("Elapsed Time:  %3.5f ms \n", time);


	CUDACHECKERROR(hipFree(vertexList));
	CUDACHECKERROR(hipFree(edgeList));
	CUDACHECKERROR(hipFree(frontierList));
	CUDACHECKERROR(hipFree(visitedList));
	CUDACHECKERROR(hipFree(distanceList));
	CUDACHECKERROR(hipFree(pathList));

}